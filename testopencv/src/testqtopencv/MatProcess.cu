#include "hip/hip_runtime.h"
#ifndef _MATCHPROCESS_CU_
#define _MATCHPROCESS_CU_
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudafilters.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "CUDACache.h"

/*
cv::Mat gaussianMat, tmpMat, tmpMat2;
tmpMat = (m - origin + 128);

cv::GaussianBlur(tmpMat, gaussianMat, cv::Size(w, h), sigmaX, sigmaY);
tmpMat = origin + 2 * gaussianMat - 256;
m = (origin * (100 - opacity) + tmpMat * opacity) / 100;
*/

__device__
uchar saturate_cast(int v)
{
	return (uchar)((unsigned)v <= UCHAR_MAX ? v : v > 0 ? UCHAR_MAX : 0);
}

__global__
void doGaussianGPUFirst(const uchar *origin, const uchar *processed, uchar *result, int width, int height, int channels, int bytesPerLine)
{
	//tmpMat = (m - origin + 128);
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= height)
	{
		return;
	}
	const uchar *orginTmp = origin + (bytesPerLine * row);
	const uchar *processedTmp = processed + (bytesPerLine * row);
	uchar *resultTmp = result + (bytesPerLine * row);
	for (int i = 0; i < width * channels; i++)
	{
		resultTmp[i] = saturate_cast((int)processedTmp[i] - (int)orginTmp[i] + 128);
	}
}

__global__
void doGaussianGPUSecond(const uchar *origin, const uchar *gaussian, uchar *result, int width, int height, int channels, int bytesPerLine, int opacity)
{
	/*
	tmpMat = origin + 2 * gaussianMat - 256;
	m = (origin * (100 - opacity) + tmpMat * opacity) / 100;
	*/
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= height)
	{
		return;
	}
	const uchar *orginTmp = origin + (bytesPerLine * row);
	const uchar *gaussianTmp = gaussian + (bytesPerLine * row);
	uchar *resultTmp = result + (bytesPerLine * row);
	for (int i = 0; i < width * channels; i++)
	{
		int v = ((int)orginTmp[i] * (100 - opacity) + ((int)orginTmp[i] + 2 * (int)gaussianTmp[i] - 256) * opacity) / 100;
		resultTmp[i] = saturate_cast(v);
	}
}

extern CUDACache g_CUDACache;

#define BLOCK_SIZE 32 * 2

extern "C" void doGaussianGPU(const cv::cuda::GpuMat & origin, cv::cuda::GpuMat & m, int w, int h, double sigmaX, double sigmaY, int opacity)
{
	cv::cuda::GpuMat tmpMat(m.size(), m.type(), cv::Scalar(1,1,1)), gaussianMat(m.size(), m.type(), cv::Scalar(0, 0, 0));

	int width = origin.cols, height = origin.rows;
	int channels = origin.channels();
	int bytesPerLine = origin.step;

	const int blockSize = BLOCK_SIZE;
	int gridSize = height / blockSize;
	if (height % blockSize != 0)
	{
		gridSize++;
	}
	//tmpMat = (m - origin + 128);
	doGaussianGPUFirst << <gridSize, blockSize >> > (origin.data, m.data, tmpMat.data, width, height, channels, bytesPerLine);
	checkCudaErrors(hipDeviceSynchronize());

	/*cv::Mat tt;
	tmpMat.download(tt);
	cv::imshow("tt", tt);*/

	//cv::GaussianBlur(tmpMat, gaussianMat, cv::Size(w, h), sigmaX, sigmaY);
	auto GaussianBlur = g_CUDACache.createGaussianFilter(tmpMat.type(), gaussianMat.type(), cv::Size(w, h), sigmaX, sigmaY);
	if (!GaussianBlur)
	{
		return;
	}
	GaussianBlur->apply(tmpMat, gaussianMat);

	/*
	tmpMat = origin + 2 * gaussianMat - 256;
	m = (origin * (100 - opacity) + tmpMat * opacity) / 100;
	*/
	doGaussianGPUSecond << <gridSize, blockSize >> > (origin.data, gaussianMat.data, m.data, width, height, channels, bytesPerLine, opacity);
	checkCudaErrors(hipDeviceSynchronize());
}

__device__
void cudargb2yuv(uchar r, uchar g, uchar b, uchar *y, uchar *u, uchar *v)
{
	*y = 0.299f * r + 0.587f * g + 0.114f * b;
	*u = -0.169f * r - 0.331f * g + 0.5f * b + 128;
	*v = 0.5f * r - 0.419f * g - 0.081f * b + 128;
}

__device__
void cudayuv2rgb(uchar y, uchar u, uchar v, uchar * r, uchar * g, uchar * b)
{
	int tmp = y + 1.13983 * (v - 128);
	*r = saturate_cast(tmp);

	tmp = y - 0.39465 * (u - 128) - 0.58060 * (v - 128);
	*g = saturate_cast(tmp);

	tmp = y + 2.03211 * (u - 128);
	*b = saturate_cast(tmp);
}

__global__
void makeWhiteGPU(uchar *data, int width, int height, int channels, int bytesPerLine, const int *mapTable)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= height)
	{
		return;
	}
	uchar *dataTmp = data + (row * bytesPerLine);
	uchar y, u, v;
	
#if 0
	for (int i = 0; i < width * channels; i += 3)
	{
		cudargb2yuv(dataTmp[i + 2], dataTmp[i + 1], dataTmp[i + 0], &y, &u, &v);
		y = saturate_cast(mapTable[y]);
		cudayuv2rgb(y, u, v, &(dataTmp[i + 2]), &(dataTmp[i + 1]), &(dataTmp[i + 0]));
	}
#else
	for (int i = 0; i < width * channels; i++)
	{
		dataTmp[i] = mapTable[dataTmp[i]];
	}
#endif
	
}

extern "C" void doMakeWhiteGPU(cv::cuda::GpuMat & origin, const int *mapTable)
{
	int width = origin.cols, height = origin.rows;
	int channels = origin.channels();
	int bytesPerLine = origin.step;
	const int blockSize = BLOCK_SIZE;
	int gridSize = height / blockSize;
	if (height % blockSize != 0)
	{
		gridSize++;
	}
	
	makeWhiteGPU << <gridSize, blockSize >> > (origin.data, width, height, channels, bytesPerLine, mapTable);
	checkCudaErrors(hipDeviceSynchronize());
}
#endif