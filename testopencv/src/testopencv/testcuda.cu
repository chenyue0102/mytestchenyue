#include "hip/hip_runtime.h"
#ifndef _TESTCUDA_CU_
#define _TESTCUDA_CU_
// CUDA utilities and system includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions

__global__ 
void fill(int *pc, int width, int height, int totalThreads)
{
	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int *tmp = pc + (width* y);
	for (int x = 0; x < width; x++)
	{
		tmp[x] = (y << 16) + x;
	}
	y += totalThreads;
	if (y < height)
	{
		tmp = pc + (width* y);
		for (int x = 0; x < width; x++)
		{
			tmp[x] = (y << 16) + x;
		}
	}
}
extern "C" void testcuda(int width, int height, int nthreads)
{
	int *p = new int[width * height]();
	int *pc = 0;
	checkCudaErrors(hipMalloc(&pc, sizeof(int) * width * height));
	checkCudaErrors(hipMemcpy(pc, p, sizeof(int) * width * height, hipMemcpyHostToDevice));

	int totalThreads = height / nthreads * nthreads;
	fill << <height / nthreads, nthreads, 0 >> > (pc, width, height, totalThreads);

	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(p, pc, sizeof(int)* width * height, hipMemcpyDeviceToHost);

	for (int y = 0; y < height; y++)
	{
		int *tmp = p + (width * y);
		for (int x = 0; x < width; x++)
		{
			int o = (y << 16) + x;
			int r = tmp[x];
			if (o != r)
			{
				printf("error x=%d y=%d r=%d", x, y, r);
			}
		}
	}

	hipFree(pc);
	delete[]p;
}
#endif