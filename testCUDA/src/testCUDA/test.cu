#include "hip/hip_runtime.h"
#ifndef _TEST_CU_
#define _TEST_CU_
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__device__
__host__
void calcValue(double *f, int width, int height, int row)
{
	f += row * width;
	for (int i = 0; i < width; i++)
	{
		for (int c = 0; c < 600; c++)
		{
			f[i] = std::pow(f[i], 2);
			f[i] = std::sqrt(f[i]);
		}
	}
}
__global__
void cudaCalc(double *f, int width, int height)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= height)
	{
		return;
	}
	calcValue(f, width, height, row);
}
#define BLOCK_SIZE 32 * 2

extern "C" void testCUDA1(double *f, int width, int height)
{
	const int blockSize = BLOCK_SIZE;
	int gridSize = height / blockSize;
	if (height % blockSize != 0)
	{
		gridSize++;
	}
	double *fCuda = nullptr;
	int byteCount = sizeof(double) * width * height;
	checkCudaErrors(hipMalloc(&fCuda, byteCount));
	checkCudaErrors(hipMemcpy(fCuda, f, byteCount, hipMemcpyHostToDevice));
	cudaCalc << <gridSize, blockSize >> > (fCuda, width, height);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(f, fCuda, byteCount, hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(fCuda));
}

extern "C" void testCUDA2(double *d, int width, int height)
{
	const int blockSize = BLOCK_SIZE;
	int gridSize = height / blockSize;
	if (height % blockSize != 0)
	{
		gridSize++;
	}
	cudaCalc << <gridSize, blockSize >> > (d, width, height);
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C" void testCPU(double *f, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		calcValue(f, width, height, i);
	}
}

__global__
void fillNumber(long long  *p)
{
	int blockId = blockIdx.x + (gridDim.x * blockIdx.y) + (gridDim.x * gridDim.y * blockIdx.z);
	int offset = threadIdx.x + (threadIdx.y * blockDim.x) + (blockDim.x * blockDim.y * threadIdx.z) + (blockId * blockDim.x * blockDim.y * blockDim.z);
	int item[] = { blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x };
	long long  value = 0;
	
	for (int i : item)
	{
		value += i;
		value *= 10;
	}
	value /= 10;
	p[offset] = value;
}

extern "C" void testGridBlock()
{
	long long data[2][3][4][5][6][7] = { 0 };
	long long  *p = 0;
	hipMallocManaged(&p, sizeof(data));
	dim3 grid(4, 3, 2), block(7,6,5);
	fillNumber << <grid, block >> > (p);
	checkCudaErrors(hipDeviceSynchronize());
	memcpy(data, p, sizeof(data));
}




extern "C" void testYUV2RGB()
{

}
#endif